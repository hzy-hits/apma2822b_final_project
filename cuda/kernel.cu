#include "hip/hip_runtime.h"
#include "kernel.cuh"

__global__ void helloKernel()
{
    printf("Hello World from GPU!\n");
}

void helloFromGPU()
{

    helloKernel<<<1, 10>>>();
    hipDeviceSynchronize();
}

// __global__ void sparseMatrixVecDotKernel(const float *val, const int *colInd, const int *indexPtr,
//                                          const float *vec, float *result, int startRow, int numRows, int vecSize)
// {
//     int row = blockIdx.x * blockDim.x + threadIdx.x + startRow;
//     // extern __shared__ float sharedVec[];

//     // for (int i = threadIdx.x; i < vecSize; i += blockDim.x)
//     // {
//     //     sharedVec[i] = vec[i];
//     // }
//     // __syncthreads();

//     if (row < numRows)
//     {
//         float dotProduct = 0.0f;
//         for (int j = indexPtr[row]; j < indexPtr[row + 1]; j++)
//         {
//             dotProduct += val[j] * vec[colInd[j]];
//         }
//         result[row - startRow] = dotProduct;
//     }
// }

__global__ void sparseMatrixVecDotKernel(const float *val, const int *colInd, const int *indexPtr,
                                         const float *vec, float *result, int numRows, int vecSize)
{
    int row = blockIdx.x * blockDim.x + threadIdx.x;

    // 使用共享内存来存储 vec 向量（如果大小适合）
    // extern __shared__ float sharedVec[];
    // if (threadIdx.x < vecSize)
    // {
    //     sharedVec[threadIdx.x] = vec[threadIdx.x];
    // }
    // __syncthreads(); // 确保所有数据都加载到 sharedVec

    if (row < numRows)
    {
        float dotProduct = 0.0f;
        for (int j = indexPtr[row]; j < indexPtr[row + 1]; j++)
        {
            dotProduct += val[j] * vec[colInd[j]];
        }
        result[row] = dotProduct;
    }
}
void kernelSparseMatVecdot(const std::vector<float> &val,
                           const std::vector<int> &colInd,
                           const std::vector<int> &indexPtr,
                           const std::vector<float> &vec,
                           std::vector<float> &result)
{
    // Check for valid inputs (omitted for brevity)

    // Allocate memory on GPU
    float *d_val, *d_vec, *d_result;
    int *d_colInd, *d_indexPtr;
    int sharedMemPerBlock;
    hipDeviceGetAttribute(&sharedMemPerBlock, hipDeviceAttributeMaxSharedMemoryPerBlock, 0);

    // Use hipMalloc to allocate memory (omitted for brevity)
    hipMalloc(&d_val, val.size() * sizeof(float));
    hipMalloc(&d_colInd, colInd.size() * sizeof(int));
    hipMalloc(&d_indexPtr, indexPtr.size() * sizeof(int));
    hipMalloc(&d_vec, vec.size() * sizeof(float));
    hipMalloc(&d_result, result.size() * sizeof(float));
    // Copy data to GPU

    hipMemcpy(d_val, val.data(), val.size() * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_colInd, colInd.data(), colInd.size() * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_indexPtr, indexPtr.data(), indexPtr.size() * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_vec, vec.data(), vec.size() * sizeof(float), hipMemcpyHostToDevice);
    auto sizeVec = vec.size();
    // Use hipMemcpy (omitted for brevity)

    // Calculate grid and block sizes
    int blockSize = 256; // Example block size, adjust as needed
    int numBlocks = (indexPtr.size() + blockSize - 1) / blockSize;

    // Launch kernel
    auto start = std::chrono::steady_clock::now();
    sparseMatrixVecDotKernel<<<numBlocks, blockSize>>>(d_val, d_colInd, d_indexPtr, d_vec, d_result, indexPtr.size() - 1, vec.size());

    // Copy results back to host
    hipMemcpy(result.data(), d_result, result.size() * sizeof(float), hipMemcpyDeviceToHost);

    auto end = std::chrono::steady_clock::now();
    printf("kernelSparseMatVecdot time: %f ms\n", std::chrono::duration<double, std::milli>(end - start).count());
    // Clean up, free GPU memory
    hipFree(d_val);
    hipFree(d_colInd);
    hipFree(d_indexPtr);
    hipFree(d_vec);
    hipFree(d_result);
}
// void kernelSparseMatVecdot(const std::vector<float> &val,
//                            const std::vector<int> &colInd,
//                            const std::vector<int> &indexPtr,
//                            const std::vector<float> &vec,
//                            std::vector<float> &result)
// {
//     // 省略输入检查和内存分配代码
//     float *d_val, *d_vec, *d_result;
//     int *d_colInd, *d_indexPtr;
//     // Use hipMalloc to allocate memory (omitted for brevity)
//     hipMalloc(&d_val, val.size() * sizeof(float));
//     hipMalloc(&d_colInd, colInd.size() * sizeof(int));
//     hipMalloc(&d_indexPtr, indexPtr.size() * sizeof(int));
//     hipMalloc(&d_vec, vec.size() * sizeof(float));
//     hipMalloc(&d_result, result.size() * sizeof(float));
//     // Copy data to GPU
//     hipMemcpy(d_val, val.data(), val.size() * sizeof(float), hipMemcpyHostToDevice);
//     hipMemcpy(d_colInd, colInd.data(), colInd.size() * sizeof(int), hipMemcpyHostToDevice);
//     hipMemcpy(d_indexPtr, indexPtr.data(), indexPtr.size() * sizeof(int), hipMemcpyHostToDevice);
//     hipMemcpy(d_vec, vec.data(), vec.size() * sizeof(float), hipMemcpyHostToDevice);
//     auto sizeVec = vec.size();
//     // 初始化 cuSPARSE
//     hipsparseHandle_t handle;
//     hipsparseCreate(&handle);
//     // 准备矩阵描述符
//     hipsparseSpMatDescr_t matA;
//     hipsparseCreateCsr(&matA, indexPtr.size() - 1, vec.size(), val.size(),
//                       d_indexPtr, d_colInd, d_val,
//                       HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I,
//                       HIPSPARSE_INDEX_BASE_ZERO, HIP_R_32F);

//     // 准备向量描述符
//     hipsparseDnVecDescr_t vecX, vecY;
//     hipsparseCreateDnVec(&vecX, vec.size(), d_vec, HIP_R_32F);
//     hipsparseCreateDnVec(&vecY, indexPtr.size() - 1, d_result, HIP_R_32F);

//     // 执行矩阵向量乘法
//     float alpha = 1.0f;
//     float beta = 0.0f;
//     void *dBuffer = NULL;
//     size_t bufferSize = 0;
//     hipsparseSpMV_bufferSize(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE,
//                             &alpha, matA, vecX, &beta, vecY, HIP_R_32F,
//                             HIPSPARSE_MV_ALG_DEFAULT, &bufferSize);
//     hipMalloc(&dBuffer, bufferSize);
//     hipsparseSpMV(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE,
//                  &alpha, matA, vecX, &beta, vecY, HIP_R_32F,
//                  HIPSPARSE_MV_ALG_DEFAULT, dBuffer);
//     hipGetLastError();
//     hipDeviceSynchronize();
//     // 将结果复制回主机
//     hipMemcpy(result.data(), d_result, result.size() * sizeof(float), hipMemcpyDeviceToHost);

//     // 清理资源
//     hipsparseDestroySpMat(matA);
//     hipsparseDestroyDnVec(vecX);
//     hipsparseDestroyDnVec(vecY);
//     hipsparseDestroy(handle);
//     // 省略释放 GPU 内存的代码
//     hipFree(d_val);
//     hipFree(d_colInd);
//     hipFree(d_indexPtr);
//     hipFree(d_vec);
//     hipFree(d_result);
// }
// void kernelSparseMatVecdot(const std::vector<float> &val,
//                            const std::vector<int> &colInd,
//                            const std::vector<int> &indexPtr,
//                            const std::vector<float> &vec,
//                            std::vector<float> &result)
// {
//     int M = 4;
//     int blockSize = 256;
//     int numRows = indexPtr.size() - 1;
//     int numBlocks = (numRows + blockSize - 1) / blockSize;

//     int rowsPerStream = numRows / M;
//     int remainingRows = numRows % M;

//     // 创建 CUDA 流
//     hipStream_t streams[M];
//     for (int i = 0; i < M; ++i)
//     {
//         hipStreamCreate(&streams[i]);
//     }

//     // 在设备上分配内存
//     float *d_val, *d_vec, *d_result;
//     int *d_colInd, *d_indexPtr;
//     hipMalloc(&d_val, val.size() * sizeof(float));
//     hipMalloc(&d_colInd, colInd.size() * sizeof(int));
//     hipMalloc(&d_indexPtr, indexPtr.size() * sizeof(int));
//     hipMalloc(&d_vec, vec.size() * sizeof(float));
//     hipMalloc(&d_result, numRows * sizeof(float));
//     hipHostRegister(const_cast<float *>(val.data()), val.size() * sizeof(float), hipHostRegisterDefault);
//     hipHostRegister(const_cast<int *>(colInd.data()), colInd.size() * sizeof(int), hipHostRegisterDefault);
//     hipHostRegister(const_cast<int *>(indexPtr.data()), indexPtr.size() * sizeof(int), hipHostRegisterDefault);
//     hipHostRegister(const_cast<float *>(vec.data()), vec.size() * sizeof(float), hipHostRegisterDefault);
//     hipHostRegister(result.data(), vec.size() * sizeof(float), hipHostRegisterDefault);
//     auto sizeVec = vec.size();
//     // 将数据拷贝到设备
//     hipMemcpy(d_vec, vec.data(), vec.size() * sizeof(float), hipMemcpyHostToDevice);

//     for (int i = 0; i < M; ++i)
//     {
//         int startRow = i * rowsPerStream;
//         int endRow = (i != M - 1) ? startRow + rowsPerStream : numRows;

//         size_t valStart = indexPtr[startRow];
//         size_t valEnd = indexPtr[endRow];
//         hipMemcpyAsync(d_val + valStart, val.data() + valStart, (valEnd - valStart) * sizeof(float), hipMemcpyHostToDevice, streams[i]);
//         hipMemcpyAsync(d_colInd + valStart, colInd.data() + valStart, (valEnd - valStart) * sizeof(int), hipMemcpyHostToDevice, streams[i]);
//         hipMemcpyAsync(d_indexPtr + startRow, indexPtr.data() + startRow, (endRow - startRow + 1) * sizeof(int), hipMemcpyHostToDevice, streams[i]);

//         sparseMatrixVecDotKernel<<<numBlocks / M + 1, blockSize, 0, streams[i]>>>(d_val, d_colInd, d_indexPtr, d_vec, d_result + startRow, startRow, endRow, sizeVec);
//         // sizeVec * sizeof(float)
//         hipMemcpyAsync(result.data() + startRow, d_result + startRow, (endRow - startRow) * sizeof(float), hipMemcpyDeviceToHost, streams[i]);
//     }

//     // 等待所有流完成
//     for (int i = 0; i < M; ++i)
//     {
//         hipStreamSynchronize(streams[i]);
//         hipStreamDestroy(streams[i]);
//     }

//     // 释放设备内存
//     hipHostUnregister(const_cast<float *>(val.data()));
//     hipHostUnregister(const_cast<int *>(colInd.data()));
//     hipHostUnregister(const_cast<int *>(indexPtr.data()));
//     hipHostUnregister(const_cast<float *>(vec.data()));
//     hipHostUnregister(result.data());
//     hipFree(d_val);
//     hipFree(d_colInd);
//     hipFree(d_indexPtr);
//     hipFree(d_vec);
//     hipFree(d_result);
// }
