#include "hip/hip_runtime.h"
#include "kernel.cuh"

__global__ void helloKernel()
{
    printf("Hello World from GPU!\n");
}

void helloFromGPU()
{

    helloKernel<<<1, 10>>>();
    hipDeviceSynchronize();
}

// __global__ void sparseMatrixVecDotKernel(const float *val, const int *colInd, const int *indexPtr,
//                                          const float *vec, float *result, int startRow, int numRows, int vecSize)
// {
//     int row = blockIdx.x * blockDim.x + threadIdx.x + startRow;
//     // extern __shared__ float sharedVec[];

//     // for (int i = threadIdx.x; i < vecSize; i += blockDim.x)
//     // {
//     //     sharedVec[i] = vec[i];
//     // }
//     // __syncthreads();

//     if (row < numRows)
//     {
//         float dotProduct = 0.0f;
//         for (int j = indexPtr[row]; j < indexPtr[row + 1]; j++)
//         {
//             dotProduct += val[j] * vec[colInd[j]];
//         }
//         result[row - startRow] = dotProduct;
//     }
// }

__global__ void sparseMatrixVecDotKernel(const float *val, const int *colInd, const int *indexPtr,
                                         const float *vec, float *result, int numRows, int vecSize)
{
    int row = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < numRows)
    {
        float dotProduct = 0.0f;
        for (int j = indexPtr[row]; j < indexPtr[row + 1]; j++)
        {
            dotProduct += val[j] * vec[colInd[j]];
        }
        result[row] = dotProduct;
    }
}
void kernelSparseMatVecdot(const std::vector<float> &val,
                           const std::vector<int> &colInd,
                           const std::vector<int> &indexPtr,
                           const std::vector<float> &vec,
                           std::vector<float> &result)
{
    // Check for valid inputs (omitted for brevity)

    // Allocate memory on GPU
    float *d_val, *d_vec, *d_result;
    int *d_colInd, *d_indexPtr;

    // Use hipMalloc to allocate memory (omitted for brevity)
    hipMalloc(&d_val, val.size() * sizeof(float));
    hipMalloc(&d_colInd, colInd.size() * sizeof(int));
    hipMalloc(&d_indexPtr, indexPtr.size() * sizeof(int));
    hipMalloc(&d_vec, vec.size() * sizeof(float));
    hipMalloc(&d_result, result.size() * sizeof(float));
    // Copy data to GPU
    hipMemcpy(d_val, val.data(), val.size() * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_colInd, colInd.data(), colInd.size() * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_indexPtr, indexPtr.data(), indexPtr.size() * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_vec, vec.data(), vec.size() * sizeof(float), hipMemcpyHostToDevice);
    auto sizeVec = vec.size();
    // Use hipMemcpy (omitted for brevity)

    // Calculate grid and block sizes
    int blockSize = 256; // Example block size, adjust as needed
    int numBlocks = (indexPtr.size() + blockSize - 1) / blockSize;

    // Launch kernel
    sparseMatrixVecDotKernel<<<numBlocks, blockSize>>>(d_val, d_colInd, d_indexPtr, d_vec, d_result, indexPtr.size(), vec.size());

    // Copy results back to host
    hipMemcpy(result.data(), d_result, result.size() * sizeof(float), hipMemcpyDeviceToHost);

    // Clean up, free GPU memory
    hipFree(d_val);
    hipFree(d_colInd);
    hipFree(d_indexPtr);
    hipFree(d_vec);
    hipFree(d_result);
}

// void kernelSparseMatVecdot(const std::vector<float> &val,
//                            const std::vector<int> &colInd,
//                            const std::vector<int> &indexPtr,
//                            const std::vector<float> &vec,
//                            std::vector<float> &result)
// {
//     int M = 4;
//     int blockSize = 256;
//     int numRows = indexPtr.size() - 1;
//     int numBlocks = (numRows + blockSize - 1) / blockSize;

//     int rowsPerStream = numRows / M;
//     int remainingRows = numRows % M;

//     // 创建 CUDA 流
//     hipStream_t streams[M];
//     for (int i = 0; i < M; ++i)
//     {
//         hipStreamCreate(&streams[i]);
//     }

//     // 在设备上分配内存
//     float *d_val, *d_vec, *d_result;
//     int *d_colInd, *d_indexPtr;
//     hipMalloc(&d_val, val.size() * sizeof(float));
//     hipMalloc(&d_colInd, colInd.size() * sizeof(int));
//     hipMalloc(&d_indexPtr, indexPtr.size() * sizeof(int));
//     hipMalloc(&d_vec, vec.size() * sizeof(float));
//     hipMalloc(&d_result, numRows * sizeof(float));
//     hipHostRegister(const_cast<float *>(val.data()), val.size() * sizeof(float), hipHostRegisterDefault);
//     hipHostRegister(const_cast<int *>(colInd.data()), colInd.size() * sizeof(int), hipHostRegisterDefault);
//     hipHostRegister(const_cast<int *>(indexPtr.data()), indexPtr.size() * sizeof(int), hipHostRegisterDefault);
//     hipHostRegister(const_cast<float *>(vec.data()), vec.size() * sizeof(float), hipHostRegisterDefault);
//     hipHostRegister(result.data(), vec.size() * sizeof(float), hipHostRegisterDefault);
//     auto sizeVec = vec.size();
//     // 将数据拷贝到设备
//     hipMemcpy(d_vec, vec.data(), vec.size() * sizeof(float), hipMemcpyHostToDevice);

//     for (int i = 0; i < M; ++i)
//     {
//         int startRow = i * rowsPerStream;
//         int endRow = (i != M - 1) ? startRow + rowsPerStream : numRows;

//         size_t valStart = indexPtr[startRow];
//         size_t valEnd = indexPtr[endRow];
//         hipMemcpyAsync(d_val + valStart, val.data() + valStart, (valEnd - valStart) * sizeof(float), hipMemcpyHostToDevice, streams[i]);
//         hipMemcpyAsync(d_colInd + valStart, colInd.data() + valStart, (valEnd - valStart) * sizeof(int), hipMemcpyHostToDevice, streams[i]);
//         hipMemcpyAsync(d_indexPtr + startRow, indexPtr.data() + startRow, (endRow - startRow + 1) * sizeof(int), hipMemcpyHostToDevice, streams[i]);

//         sparseMatrixVecDotKernel<<<numBlocks / M + 1, blockSize, 0, streams[i]>>>(d_val, d_colInd, d_indexPtr, d_vec, d_result + startRow, startRow, endRow, sizeVec);
//         // sizeVec * sizeof(float)
//         hipMemcpyAsync(result.data() + startRow, d_result + startRow, (endRow - startRow) * sizeof(float), hipMemcpyDeviceToHost, streams[i]);
//     }

//     // 等待所有流完成
//     for (int i = 0; i < M; ++i)
//     {
//         hipStreamSynchronize(streams[i]);
//         hipStreamDestroy(streams[i]);
//     }

//     // 释放设备内存
//     hipHostUnregister(const_cast<float *>(val.data()));
//     hipHostUnregister(const_cast<int *>(colInd.data()));
//     hipHostUnregister(const_cast<int *>(indexPtr.data()));
//     hipHostUnregister(const_cast<float *>(vec.data()));
//     hipHostUnregister(result.data());
//     hipFree(d_val);
//     hipFree(d_colInd);
//     hipFree(d_indexPtr);
//     hipFree(d_vec);
//     hipFree(d_result);
// }
